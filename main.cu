
#include <hip/hip_runtime.h>
#include<cstdio>
#include<cstdlib>

__global__ void print_from_gpu() {
    printf("Hello World! from thread [%d,%d] From device\n", threadIdx.x,blockIdx.x);
}

int main() {
    printf("Hello World from host!\n");
    print_from_gpu<<<1, 1>>>();
    hipDeviceSynchronize();

    return EXIT_SUCCESS;
}
